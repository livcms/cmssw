#include "hip/hip_runtime.h"
#include "test_CUDAScopedContextKernels.h"

namespace {
  __global__ void single_mul(int *d) { d[0] = d[0] * 2; }

  __global__ void join_add(const int *d1, const int *d2, int *d3) { d3[0] = d1[0] + d2[0]; }
}  // namespace

void testCUDAScopedContextKernels_single(int *d, hipStream_t stream) { single_mul<<<1, 1, 0, stream>>>(d); }

void testCUDAScopedContextKernels_join(const int *d1, const int *d2, int *d3, hipStream_t stream) {
  join_add<<<1, 1, 0, stream>>>(d1, d2, d3);
}
